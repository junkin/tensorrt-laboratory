
#include "cuda_mel_banks.h"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <iostream>
#include <sstream>
#include <vector>
#include <string>
#include <iterator>

CudaMelBanks::CudaMelBanks(const Config& config)
: MelBanks(config)
{
    const auto& bins = MelBanks::GetBins();
    auto        size = bins.size();

    h_vecs.resize(size);
    h_sizes.resize(size);
    h_offsets.resize(size);
    d_vecs.resize(size);



    for (int i = 0; i < bins.size(); i++)
    {
        d_vecs[i].Resize(bins[i].second.size(), kUndefined);
        d_vecs[i].CopyFromVec(bins[i].second);
        h_vecs[i]    = d_vecs[i].Data();
        h_sizes[i]   = d_vecs[i].Dim();
        h_offsets[i] = bins[i].first;
    }

    d_sizes.CopyFromVec(h_sizes);
    d_offsets.CopyFromVec(h_offsets);

    CHECK_EQ(hipMalloc((void**)&d_vecs_ptrs, size * sizeof(float*)), hipSuccess);
    CHECK_EQ(hipMemcpyAsync(d_vecs_ptrs, &h_vecs[0], size * sizeof(float*), hipMemcpyHostToDevice, hipStreamPerThread), hipSuccess);
    CHECK_EQ(hipStreamSynchronize(hipStreamPerThread), hipSuccess);
}

CudaMelBanks::~CudaMelBanks()
{
    CHECK_EQ(hipStreamSynchronize(hipStreamPerThread), hipSuccess);
    CHECK_EQ(hipFree(d_vecs_ptrs), hipSuccess);

}
